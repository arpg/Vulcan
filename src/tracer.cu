#include "hip/hip_runtime.h"
#include <vulcan/tracer.h>
#include <vulcan/tracer.cuh>
#include <cfloat>
#include <vulcan/hash.h>
#include <vulcan/projection.h>
#include <vulcan/transform.h>
#include <vulcan/util.cuh>
#include <vulcan/voxel.h>

namespace vulcan
{

template <int BLOCK_SIZE>
VULCAN_GLOBAL
void ComputePatchesKernel(const int* indices, const HashEntry* entries,
    const Transform Tcw, const Projection projection, float block_length,
    int block_count, int image_width, int image_height, int bounds_width,
    int bounds_height, Patch* patches, int* patch_count)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;

  Vector2s bmax(-1, -1);
  Vector2s bmin(bounds_width, bounds_height);
  Vector2f depth_bounds(+FLT_MAX, -FLT_MAX);
  const int patch_size = Patch::max_size;

  if (index < block_count)
  {
    const int entry_index = indices[index];
    const HashEntry entry = entries[entry_index];
    const Block& block = entry.block;
    const Vector3s& origin = block.GetOrigin();
    Vector4f Xwp(0, 0, 0, 1);

    for (int z = 0; z <= 1; ++z)
    {
      Xwp[2] = block_length * (z + origin[2]);

      for (int y = 0; y <= 1; ++y)
      {
        Xwp[1] = block_length * (y + origin[1]);

        for (int x = 0; x <= 1; ++x)
        {
          Xwp[0] = block_length * (x + origin[0]);

          const Vector3f Xcp = Vector3f(Tcw * Xwp);
          Vector2f uv = projection.Project(Xcp);

          uv[0] = bounds_width * uv[0] / image_width;
          uv[1] = bounds_height * uv[1] / image_height;

          bmin[0] = clamp<short>(min((short)floorf(uv[0]), bmin[0]), 0, bounds_width - 1);
          bmin[1] = clamp<short>(min((short)floorf(uv[1]), bmin[1]), 0, bounds_height - 1);

          bmax[0] = clamp<short>(max((short)ceilf(uv[0]), bmax[0]), 0, bounds_width - 1);
          bmax[1] = clamp<short>(max((short)ceilf(uv[1]), bmax[1]), 0, bounds_height - 1);

          depth_bounds[0] = min(Xcp[2], depth_bounds[0]);
          depth_bounds[1] = max(Xcp[2], depth_bounds[1]);
        }
      }
    }
  }

  const int rx = max(0, bmax[0] - bmin[0]);
  const int ry = max(0, bmax[1] - bmin[1]);
  const int gx = (rx + patch_size - 1) / patch_size;
  const int gy = (ry + patch_size - 1) / patch_size;
  const int count = gx * gy;

  const int offset = PrefixSum<BLOCK_SIZE>(count, threadIdx.x, *patch_count);

  for (int i = 0; i < gy; ++i)
  {
    for (int j = 0; j < gx; ++j)
    {
      Patch patch;
      const int output = offset + i * gx + j;
      patch.origin = bmin + patch_size * Vector2s(j, i);
      patch.size[0] = min(patch_size, bmax[0] - patch.origin[0] + 1);
      patch.size[1] = min(patch_size, bmax[1] - patch.origin[1] + 1);
      patch.bounds = depth_bounds;
      patches[output] = patch;
    }
  }
}

VULCAN_GLOBAL
void ComputeBoundsKernel(const Patch* patches, Vector2f* bounds,
    int bounds_width, int patch_count)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < patch_count)
  {
    const Patch patch = patches[index];

    for (int i = 0; i < patch.size[1]; ++i)
    {
      const int y = patch.origin[1] + i;

      for (int j = 0; j < patch.size[0]; ++j)
      {
        const int x = patch.origin[0] + j;
        const int pixel = y * bounds_width + x;
        atomicMin(&bounds[pixel][0], patch.bounds[0]);
        atomicMax(&bounds[pixel][1], patch.bounds[1]);
      }
    }
  }
}

VULCAN_DEVICE
Voxel GetVoxel(int K, const HashEntry* entries, const Voxel* voxels,
    int bx, int by, int bz, int vx, int vy, int vz)
{
  if (vx < 0)
  {
    --bx;
    vx = Block::resolution + vx;
  }
  else if (vx >= Block::resolution)
  {
    ++bx;
    vx = vx - Block::resolution;
  }

  if (vy < 0)
  {
    --by;
    vy = Block::resolution + vy;
  }
  else if (vy >= Block::resolution)
  {
    ++by;
    vy = vy - Block::resolution;
  }

  if (vz < 0)
  {
    --bz;
    vz = Block::resolution + vz;
  }
  else if (vz >= Block::resolution)
  {
    ++bz;
    vz = vz - Block::resolution;
  }

  const uint32_t P1 = 73856093;
  const uint32_t P2 = 19349669;
  const uint32_t P3 = 83492791;

  const uint32_t hash_code = ((bx * P1) ^ (by * P2) ^ (bz * P3)) % K;
  HashEntry entry = entries[hash_code];
  bool found = false;

  do
  {
    if (entry.block == Block(bx, by, bz))
    {
      found = true;
      break;
    }
    else if (!entry.HasNext())
    {
      break;
    }

    entry = entries[entry.next];
  }
  while (true);

  if (found && entry.IsAllocated())
  {
    const int r = Block::resolution;
    const int rr = r * r;

    const int block_offset = Block::voxel_count * entry.data;
    const int voxel_offset = vz * rr + vy * r + vx;
    return voxels[block_offset + voxel_offset];
  }
  else
  {
    return Voxel::Empty();
  }
}

VULCAN_DEVICE
Vector4f GetInterpolatedDistance(const HashEntry* entries, const Voxel* voxels,
    int K, float block_length, float voxel_length, int bx, int by, int bz,
    const HashEntry& entry, const Vector3f& p, float sdf)
{
  const float wx = (p[0] - bx * block_length) / voxel_length;
  const float wy = (p[1] - by * block_length) / voxel_length;
  const float wz = (p[2] - bz * block_length) / voxel_length;

  const int r = Block::resolution;
  const int rr = r * r;

  const int block_offset = Block::voxel_count * entry.data;

  Vector3f color;

  const int i0x = floorf(wx - 0.5f);
  const int i0y = floorf(wy - 0.5f);
  const int i0z = floorf(wz - 0.5f);

  if (i0x >= 0 && i0y >= 0 && i0z >= 0 &&
      i0x < Block::resolution - 1 &&
      i0y < Block::resolution - 1 &&
      i0z < Block::resolution - 1)
  {
    // compute depth via trilinear interpolation
    // all samples come from current block

    Vector3f w1;
    w1[0] = wx - (i0x + 0.5f);
    w1[1] = wy - (i0y + 0.5f);
    w1[2] = wz - (i0z + 0.5f);

    Vector3f w0 = Vector3f::Ones() - w1;

    const int i000 = (i0z + 0) * rr + (i0y + 0) * r + (i0x + 0);
    const int i001 = (i0z + 0) * rr + (i0y + 0) * r + (i0x + 1);
    const int i010 = (i0z + 0) * rr + (i0y + 1) * r + (i0x + 0);
    const int i011 = (i0z + 0) * rr + (i0y + 1) * r + (i0x + 1);
    const int i100 = (i0z + 1) * rr + (i0y + 0) * r + (i0x + 0);
    const int i101 = (i0z + 1) * rr + (i0y + 0) * r + (i0x + 1);
    const int i110 = (i0z + 1) * rr + (i0y + 1) * r + (i0x + 0);
    const int i111 = (i0z + 1) * rr + (i0y + 1) * r + (i0x + 1);

    const Voxel v000 = voxels[block_offset + i000];
    const Voxel v001 = voxels[block_offset + i001];
    const Voxel v010 = voxels[block_offset + i010];
    const Voxel v011 = voxels[block_offset + i011];
    const Voxel v100 = voxels[block_offset + i100];
    const Voxel v101 = voxels[block_offset + i101];
    const Voxel v110 = voxels[block_offset + i110];
    const Voxel v111 = voxels[block_offset + i111];

    const float n000 = v000.distance;
    const float n001 = v001.distance;
    const float n010 = v010.distance;
    const float n011 = v011.distance;
    const float n100 = v100.distance;
    const float n101 = v101.distance;
    const float n110 = v110.distance;
    const float n111 = v111.distance;

    const float n00 = n000 * w0[0] + n001 * w1[0];
    const float n01 = n010 * w0[0] + n011 * w1[0];
    const float n10 = n100 * w0[0] + n101 * w1[0];
    const float n11 = n110 * w0[0] + n111 * w1[0];

    const float n0 = n00 * w0[1] + n01 * w1[1];
    const float n1 = n10 * w0[1] + n11 * w1[1];

    const Vector3f c000 = v000.color;
    const Vector3f c001 = v001.color;
    const Vector3f c010 = v010.color;
    const Vector3f c011 = v011.color;
    const Vector3f c100 = v100.color;
    const Vector3f c101 = v101.color;
    const Vector3f c110 = v110.color;
    const Vector3f c111 = v111.color;

    const Vector3f c00 = c000 * w0[0] + c001 * w1[0];
    const Vector3f c01 = c010 * w0[0] + c011 * w1[0];
    const Vector3f c10 = c100 * w0[0] + c101 * w1[0];
    const Vector3f c11 = c110 * w0[0] + c111 * w1[0];

    const Vector3f c0 = c00 * w0[1] + c01 * w1[1];
    const Vector3f c1 = c10 * w0[1] + c11 * w1[1];

    sdf = n0 * w0[2] + n1 * w1[2];
    color = c0 * w0[2] + c1 * w1[2];
  }
  else
  {
    // compute depth via trilinear interpolation
    // requires sampling from other blocks

    Vector3f w1;
    w1[0] = wx - (i0x + 0.5f);
    w1[1] = wy - (i0y + 0.5f);
    w1[2] = wz - (i0z + 0.5f);

    Vector3f w0 = Vector3f::Ones() - w1;

    const Voxel v000 = GetVoxel(K, entries, voxels, bx, by, bz, i0x + 0, i0y + 0, i0z + 0);
    const Voxel v001 = GetVoxel(K, entries, voxels, bx, by, bz, i0x + 1, i0y + 0, i0z + 0);
    const Voxel v010 = GetVoxel(K, entries, voxels, bx, by, bz, i0x + 0, i0y + 1, i0z + 0);
    const Voxel v011 = GetVoxel(K, entries, voxels, bx, by, bz, i0x + 1, i0y + 1, i0z + 0);
    const Voxel v100 = GetVoxel(K, entries, voxels, bx, by, bz, i0x + 0, i0y + 0, i0z + 1);
    const Voxel v101 = GetVoxel(K, entries, voxels, bx, by, bz, i0x + 1, i0y + 0, i0z + 1);
    const Voxel v110 = GetVoxel(K, entries, voxels, bx, by, bz, i0x + 0, i0y + 1, i0z + 1);
    const Voxel v111 = GetVoxel(K, entries, voxels, bx, by, bz, i0x + 1, i0y + 1, i0z + 1);

    const float n000 = v000.distance;
    const float n001 = v001.distance;
    const float n010 = v010.distance;
    const float n011 = v011.distance;
    const float n100 = v100.distance;
    const float n101 = v101.distance;
    const float n110 = v110.distance;
    const float n111 = v111.distance;

    const float n00 = n000 * w0[0] + n001 * w1[0];
    const float n01 = n010 * w0[0] + n011 * w1[0];
    const float n10 = n100 * w0[0] + n101 * w1[0];
    const float n11 = n110 * w0[0] + n111 * w1[0];

    const float n0 = n00 * w0[1] + n01 * w1[1];
    const float n1 = n10 * w0[1] + n11 * w1[1];

    const Vector3f c000 = v000.color;
    const Vector3f c001 = v001.color;
    const Vector3f c010 = v010.color;
    const Vector3f c011 = v011.color;
    const Vector3f c100 = v100.color;
    const Vector3f c101 = v101.color;
    const Vector3f c110 = v110.color;
    const Vector3f c111 = v111.color;

    const Vector3f c00 = c000 * w0[0] + c001 * w1[0];
    const Vector3f c01 = c010 * w0[0] + c011 * w1[0];
    const Vector3f c10 = c100 * w0[0] + c101 * w1[0];
    const Vector3f c11 = c110 * w0[0] + c111 * w1[0];

    const Vector3f c0 = c00 * w0[1] + c01 * w1[1];
    const Vector3f c1 = c10 * w0[1] + c11 * w1[1];

    sdf = n0 * w0[2] + n1 * w1[2];
    color = c0 * w0[2] + c1 * w1[2];
  }

  return Vector4f(sdf, color[0], color[1], color[2]);
}

VULCAN_GLOBAL
void ComputePointsKernel(const HashEntry* entries, const Voxel* voxels,
    const Vector2f* bounds, int block_count, float block_length,
    float voxel_length, float trunc_length, const Transform Twc,
    const Projection projection, float* depths, Vector3f* colors,
    int image_width, int image_height, int bounds_width, int bounds_height)
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < image_width && y < image_height)
  {
    const int px = bounds_width * x / image_width;
    const int py = bounds_height * y / image_height;
    const Vector2f bound = bounds[py * bounds_width + px];

    float depth = 0;
    float final_depth = 0;
    Vector3f color = Vector3f(0, 0, 0);

    if (bound[0] < bound[1])
    {
      const Vector2f uv(x + 0.5f, y + 0.5f);
      const Vector3f Xcp = projection.Unproject(uv) * bound[0];
      const Vector3f Xwp = Vector3f(Twc * Vector4f(Xcp, 1.0f));
      const Vector3f dir = Vector3f(Twc * Vector4f(Xcp, 0.0f)).Normalized();

      const uint32_t P1 = 73856093;
      const uint32_t P2 = 19349669;
      const uint32_t P3 = 83492791;
      const uint32_t K  = block_count;

      Vector3f p = Xwp;
      const Transform Tcw = Twc.Inverse();
      const int r = Block::resolution;
      const int rr = r * r;

      depth = bound[0];
      color = Vector3f(0, 0, 0);

      int iters = 0;

      do
      {
        const int bx = floorf(p[0] / block_length);
        const int by = floorf(p[1] / block_length);
        const int bz = floorf(p[2] / block_length);
        const uint32_t hash_code = ((bx * P1) ^ (by * P2) ^ (bz * P3)) % K;
        HashEntry entry = entries[hash_code];

        while (entry.block != Block(bx, by, bz) && entry.HasNext())
        {
          entry = entries[entry.next];
        }

        if (entry.block == Block(bx, by, bz) && entry.IsAllocated())
        {
          const float wx = (p[0] - bx * block_length) / voxel_length;
          const float wy = (p[1] - by * block_length) / voxel_length;
          const float wz = (p[2] - bz * block_length) / voxel_length;

          const int vx = wx;
          const int vy = wy;
          const int vz = wz;

          const int block_offset = Block::voxel_count * entry.data;
          const int voxel_offset = vz * rr + vy * r + vx;
          const Voxel voxel = voxels[block_offset + voxel_offset];
          float sdf = voxel.distance;

          if (sdf <= 0.1f && sdf >= -0.5f)
          {
            const Vector4f v = GetInterpolatedDistance(entries, voxels, K,
                block_length, voxel_length, bx, by, bz, entry, p, sdf);

            sdf = v[0];
            color = Vector3f(v[1], v[2], v[3]);
          }

          if (sdf <= 0.0f)
          {
            p += trunc_length * sdf * dir;

            const int bx = floorf(p[0] / block_length);
            const int by = floorf(p[1] / block_length);
            const int bz = floorf(p[2] / block_length);
            const uint32_t hash_code = ((bx * P1) ^ (by * P2) ^ (bz * P3)) % K;
            HashEntry entry = entries[hash_code];

            while (entry.block != Block(bx, by, bz) && entry.HasNext())
            {
              entry = entries[entry.next];
            }

            if (entry.block == Block(bx, by, bz) && entry.IsAllocated())
            {
              const Vector4f v = GetInterpolatedDistance(entries, voxels, K,
                  block_length, voxel_length, bx, by, bz, entry, p, sdf);

              sdf = v[0];
              color = Vector3f(v[1], v[2], v[3]);
              p += trunc_length * sdf * dir;
            }

            const Vector3f Xcd = Vector3f(Tcw * Vector4f(p, 1.0f));
            final_depth = Xcd[2];

            break;
          }
          else
          {
            p += max(voxel_length, trunc_length * sdf) * dir;
          }
        }
        else
        {
          p += block_length * dir;
        }

        const Vector3f Xcd = Vector3f(Tcw * Vector4f(p, 1.0f));
        depth = Xcd[2];

        if (++iters >= 100) // TODO: remove
        {
          // printf("iteration reached\n");
          color = Vector3f(1, 0, 0);
          break;
        }
        // else if (depth > bound[1])
        // {
        //   printf("depth reached\n");
        //   color = Vector3f(1, 1, 0);
        //   break;
        // }
      }
      while (depth < bound[1]);
    }

    const int pixel = y * image_width + x;
    depths[pixel] = final_depth;
    colors[pixel] = color;
  }
}

template <int PATCH_SIZE>
VULCAN_GLOBAL
void ComputeNormalsKernel(const float* depths, const Projection projection,
    Vector3f* normals, int image_width, int image_height)
{
  const int pad = 2;
  const int resolution = (PATCH_SIZE + 2 * pad);
  const int shared_size = resolution * resolution;
  VULCAN_SHARED float shared[shared_size];

  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  int shared_index = threadIdx.y * blockDim.x + threadIdx.x;

  while (shared_index < shared_size)
  {
    float depth = 0;
    const int bx = blockIdx.x * blockDim.x - pad + (shared_index % resolution);
    const int by = blockIdx.y * blockDim.y - pad + (shared_index / resolution);

    if (bx >= 0 && bx < image_width && by >= 0 && by < image_height)
    {
      depth = depths[by * image_width + bx];
    }

    shared[shared_index] = depth;
    shared_index += blockDim.x * blockDim.y;
  }

  __syncthreads();

  const float depth = shared[(threadIdx.y + pad) * resolution + (threadIdx.x + pad)];

  if (x < image_width && y < image_height)
  {
    Vector3f normal(0, 0, 0);

    if (depth > 0)
    {
      float d;
      Vector2f uv;

      uv[0] = (x + 0) + 0.5f;
      uv[1] = (y + 0) + 0.5f;
      const Vector3f z0 = projection.Unproject(uv) * depth;

      Vector3f x0;
      d = shared[(threadIdx.y + pad) * resolution + (threadIdx.x + 0)];

      if (d == 0)
      {
        x0 = z0;
      }
      else
      {
        uv[0] = (x - pad) + 0.5f;
        uv[1] = (y + 0) + 0.5f;
        x0 = projection.Unproject(uv) * d;
      }

      Vector3f x1;
      d = shared[(threadIdx.y + pad) * resolution + (threadIdx.x + 2 * pad)];

      if (d == 0)
      {
        x1 = z0;
      }
      else
      {
        uv[0] = (x + pad) + 0.5f;
        uv[1] = (y + 0) + 0.5f;
        x1 = projection.Unproject(uv) * d;
      }

      Vector3f y0;
      d = shared[(threadIdx.y + 0) * resolution + (threadIdx.x + pad)];

      if (d == 0)
      {
        y0 = z0;
      }
      else
      {
        uv[0] = (x + 0) + 0.5f;
        uv[1] = (y - pad) + 0.5f;
        y0 = projection.Unproject(uv) * d;
      }

      Vector3f y1;
      d = shared[(threadIdx.y + 2 * pad) * resolution + (threadIdx.x + pad)];

      if (d == 0)
      {
        y1 = z0;
      }
      else
      {
        uv[0] = (x + 0) + 0.5f;
        uv[1] = (y + pad) + 0.5f;
        y1 = projection.Unproject(uv) * d;
      }

      const Vector3f dx = x0 - x1;
      const Vector3f dy = y0 - y1;

      if (dx.SquaredNorm() > 0 && dy.SquaredNorm() > 0)
      {
        normal = dy.Cross(dx);
        normal.Normalize();
      }
    }

    const int output = y * image_width + x;
    normals[output] = normal;
  }
}

void ComputePatches(const int* indices, const HashEntry* entries,
    const Transform& Tcw, const Projection& projection, float block_length,
    int block_count, int image_width, int image_height, int bounds_width,
    int bounds_height, Patch* patches, int* patch_count)
{
  const int threads = 512;
  const int blocks = GetKernelBlocks(block_count, threads);

  CUDA_LAUNCH(ComputePatchesKernel<threads>, blocks, threads, 0, 0,
      indices, entries, Tcw, projection, block_length, block_count,
      image_width, image_height, bounds_width, bounds_height, patches,
      patch_count);
}

void ComputeBounds(const Patch* patches, Vector2f* bounds, int bounds_width,
    int patch_count)
{
  const int threads = 512;
  const int blocks = GetKernelBlocks(patch_count, threads);

  CUDA_LAUNCH(ComputeBoundsKernel, blocks, threads, 0, 0, patches,
      bounds, bounds_width, patch_count);
}

void ComputePoints(const HashEntry* entries, const Voxel* voxels,
    const Vector2f* bounds, int block_count, float block_length,
    float voxel_length, float trunc_length, const Transform& Twc,
    const Projection& projection, float* depths, Vector3f* colors,
    int image_width, int image_height, int bounds_width, int bounds_height)
{
  const dim3 threads(16, 16);
  const dim3 total(image_width, image_height);
  const dim3 blocks = GetKernelBlocks(total, threads);

  CUDA_LAUNCH(ComputePointsKernel, blocks, threads, 0, 0, entries, voxels,
      bounds, block_count, block_length, voxel_length, trunc_length, Twc,
      projection, depths, colors, image_width, image_height,
      bounds_width, bounds_height);
}

void ComputeNormals(const float* depths, const Projection& projection,
    Vector3f* normals, int image_width, int image_height)
{
  const dim3 threads(16, 16);
  const dim3 total(image_width, image_height);
  const dim3 blocks = GetKernelBlocks(total, threads);

  CUDA_LAUNCH(ComputeNormalsKernel<16>, blocks, threads, 0, 0, depths,
      projection, normals, image_width, image_height);
}

void ResetBoundsBuffer(Vector2f *bounds, int count)
{
  const int threads = 512;
  const int blocks = GetKernelBlocks(count, threads);
  const Vector2f value(+FLT_MAX, -FLT_MAX);
  CUDA_LAUNCH(FillKernel, blocks, threads, 0, 0, bounds, value, count);
}

} // namespace vulcan