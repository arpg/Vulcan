#include "hip/hip_runtime.h"
#include <vulcan/color_tracker.h>
#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include <thrust/host_vector.h>
#include <vulcan/device.h>
#include <vulcan/frame.h>
#include <vulcan/projection.h>
#include <vulcan/transform.h>

namespace vulcan
{

VULCAN_DEVICE
inline void WarpReduceX(volatile float* buffer, int thread)
{
  buffer[thread] += buffer[thread + 32];
  buffer[thread] += buffer[thread + 16];
  buffer[thread] += buffer[thread +  8];
  buffer[thread] += buffer[thread +  4];
  buffer[thread] += buffer[thread +  2];
  buffer[thread] += buffer[thread +  1];
}

namespace
{

VULCAN_GLOBAL
void ComputeIntensitiesKernel(int total, const Vector3f* colors,
    float* intensities)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < total)
  {
    const Vector3f color = colors[index];
    intensities[index] = (color[0] + color[1] + color[2]) / 3.0f;
  }
}

VULCAN_DEVICE
float Sample(int w, int h, const float* values, float u, float v)
{
  const int x = floorf(u - 0.5f);
  const int y = floorf(v - 0.5f);

  VULCAN_DEBUG(x >= 0 && x < w - 1 && y >= 0 && y < h - 1);

  const float v00 = values[(y + 0) * w + (x + 0)];
  const float v01 = values[(y + 0) * w + (x + 1)];
  const float v10 = values[(y + 1) * w + (x + 0)];
  const float v11 = values[(y + 1) * w + (x + 1)];

  const float u1 = u - (x + 0.5f);
  const float v1 = v - (y + 0.5f);
  const float u0 = 1.0f - u1;
  const float v0 = 1.0f - v1;

  const float w00 = v0 * u0;
  const float w01 = v0 * u1;
  const float w10 = v1 * u0;
  const float w11 = v1 * u1;

  return (w00 * v00) + (w01 * v01) + (w10 * v10) + (w11 * v11);
}

template <bool translation_enabled>
VULCAN_GLOBAL
void ComputeSystemKernel(const Transform Tcm, const float* keyframe_depths,
    const Vector3f* keyframe_normals, const float* keyframe_intensities,
    const Projection keyframe_projection, int keyframe_width,
    int keyframe_height, const float* frame_depths,
    const Vector3f* frame_normals, const float* frame_intensities,
    const float* frame_gradient_x, const float* frame_gradient_y,
    const Projection frame_projection, int frame_width, int frame_height,
    float* hessian, float* gradient, float* residuals)
{
  VULCAN_SHARED float buffer1[256];
  VULCAN_SHARED float buffer2[256];
  VULCAN_SHARED float buffer3[256];

  const int keyframe_x = blockIdx.x * blockDim.x + threadIdx.x;
  const int keyframe_y = blockIdx.y * blockDim.y + threadIdx.y;
  const int thread = threadIdx.y * blockDim.x + threadIdx.x;

  float residual = 0;
  Vector6f dfdx = Vector6f::Zeros();

  if (keyframe_x < keyframe_width && keyframe_y < keyframe_height)
  {
    const int keyframe_index = keyframe_y * keyframe_width + keyframe_x;
    const float keyframe_depth = keyframe_depths[keyframe_index];

    if (keyframe_depth > 0)
    {
      const float keyframe_u = keyframe_x + 0.5f;
      const float keyframe_v = keyframe_y + 0.5f;
      const Vector3f Xmp = keyframe_projection.Unproject(keyframe_u, keyframe_v, keyframe_depth);
      const Vector3f Xcp = Vector3f(Tcm * Vector4f(Xmp, 1));
      const Vector2f frame_uv = frame_projection.Project(Xcp);

      if (frame_uv[0] >= 0.5f && frame_uv[0] < frame_width  - 0.5f &&
          frame_uv[1] >= 0.5f && frame_uv[1] < frame_height - 0.5f)
      {
        const int frame_x = frame_uv[0];
        const int frame_y = frame_uv[1];
        const int frame_index = frame_y * frame_width + frame_x;
        const float frame_depth = frame_depths[frame_index];

        if (fabsf(frame_depth - Xcp[2]) < 0.1)
        {
          const Vector3f frame_normal = frame_normals[frame_index];
          Vector3f keyframe_normal = keyframe_normals[keyframe_index];
          keyframe_normal = Vector3f(Tcm * Vector4f(keyframe_normal, 0));

          if (keyframe_normal.SquaredNorm() > 0 &&
              frame_normal.Dot(keyframe_normal) > 0.5f)
          {
            const float Im = keyframe_intensities[keyframe_index];

            const float Ic = Sample(frame_width, frame_height,
                frame_intensities, frame_uv[0], frame_uv[1]);

            // TODO: return
            residual = Ic - Im;
            // residual = Ic;

            const float x = Xmp[0];
            const float y = Xmp[1];
            const float z = Xmp[2];

            const float fx = frame_projection.GetFocalLength()[0];
            const float fy = frame_projection.GetFocalLength()[1];

            const float gx = Sample(frame_width, frame_height, frame_gradient_x,
                frame_uv[0], frame_uv[1]);

            const float gy = Sample(frame_width, frame_height, frame_gradient_y,
                frame_uv[0], frame_uv[1]);


            dfdx[0] = -fy*gy-y*(fx*gx*x*1/(z*z)+fy*gy*y*1/(z*z));
            dfdx[1] = fx*gx+x*(fx*gx*x*1/(z*z)+fy*gy*y*1/(z*z));
            dfdx[2] = (fy*gy*x)/z-(fx*gx*y)/z;

            if (translation_enabled)
            {
              dfdx[3] = (fx*gx)/z;
              dfdx[4] = (fy*gy)/z;
              dfdx[5] = -fx*gx*x*1/(z*z)-fy*gy*y*1/(z*z);
            }
          }
        }
      }
    }

    residuals[keyframe_index] = residual; // TODO: remove
  }

  const int parameter_count = translation_enabled ? 6 : 3;

  for (int i = 0; i < parameter_count; i += 3)
  {
    buffer1[thread] = dfdx[i + 0] * residual;
    buffer2[thread] = dfdx[i + 1] * residual;
    buffer3[thread] = dfdx[i + 2] * residual;

    __syncthreads();

    if (thread < 128)
    {
      buffer1[thread] += buffer1[thread + 128];
      buffer2[thread] += buffer2[thread + 128];
      buffer3[thread] += buffer3[thread + 128];
    }

    __syncthreads();

    if (thread < 64)
    {
      buffer1[thread] += buffer1[thread + 64];
      buffer2[thread] += buffer2[thread + 64];
      buffer3[thread] += buffer3[thread + 64];
    }

    __syncthreads();

    if (thread < 32)
    {
      WarpReduceX(buffer1, thread);
      WarpReduceX(buffer2, thread);
      WarpReduceX(buffer3, thread);
    }

    if (thread == 0)
    {
      atomicAdd(&gradient[i + 0], buffer1[thread]);
      atomicAdd(&gradient[i + 1], buffer2[thread]);
      atomicAdd(&gradient[i + 2], buffer3[thread]);
    }

    __syncthreads();
  }

  const int hessian_count = translation_enabled ? 21 : 6;
  float local_hessian[hessian_count];

  for (unsigned char r = 0, counter = 0; r < parameter_count; r++)
  {
    for (int c = 0; c <= r; c++, counter++)
    {
      local_hessian[counter] = dfdx[r] * dfdx[c];
    }
  }

  for (int i = 0; i < hessian_count; i += 3)
  {
    buffer1[thread] = local_hessian[i + 0];
    buffer2[thread] = local_hessian[i + 1];
    buffer3[thread] = local_hessian[i + 2];

    __syncthreads();

    if (thread < 128)
    {
      buffer1[thread] += buffer1[thread + 128];
      buffer2[thread] += buffer2[thread + 128];
      buffer3[thread] += buffer3[thread + 128];
    }

    __syncthreads();

    if (thread < 64)
    {
      buffer1[thread] += buffer1[thread + 64];
      buffer2[thread] += buffer2[thread + 64];
      buffer3[thread] += buffer3[thread + 64];
    }

    __syncthreads();

    if (thread < 32)
    {
      WarpReduceX(buffer1, thread);
      WarpReduceX(buffer2, thread);
      WarpReduceX(buffer3, thread);
    }

    if (thread == 0)
    {
      atomicAdd(&hessian[i + 0], buffer1[thread]);
      atomicAdd(&hessian[i + 1], buffer2[thread]);
      atomicAdd(&hessian[i + 2], buffer3[thread]);
    }

    __syncthreads();
  }
}

} // namespace

void ColorTracker::ComputeKeyframeIntensities()
{
  const int width = keyframe_->depth_image->GetWidth();
  const int height = keyframe_->depth_image->GetHeight();
  keyframe_intensities_.Resize(width, height);

  const Vector3f* colors = keyframe_->color_image->GetData();
  float* intensities = keyframe_intensities_.GetData();

  const int threads = 512;
  const int total = width * height;
  const int blocks = GetKernelBlocks(total, threads);

  CUDA_LAUNCH(ComputeIntensitiesKernel, blocks, threads, 0, 0, total, colors,
      intensities);
}

void ColorTracker::ComputeFrameIntensities(const Frame& frame)
{
  const int width = frame.depth_image->GetWidth();
  const int height = frame.depth_image->GetHeight();
  frame_intensities_.Resize(width, height);

  const Vector3f* colors = frame.color_image->GetData();
  float* intensities = frame_intensities_.GetData();

  const int threads = 512;
  const int total = width * height;
  const int blocks = GetKernelBlocks(total, threads);

  CUDA_LAUNCH(ComputeIntensitiesKernel, blocks, threads, 0, 0, total, colors,
      intensities);
}

void ColorTracker::ComputeFrameGradients(const Frame& frame)
{
  frame_intensities_.GetGradients(frame_gradient_x_, frame_gradient_y_);
}

void ColorTracker::ComputeSystem(const Frame& frame)
{
  const int frame_width = frame.depth_image->GetWidth();
  const int frame_height = frame.depth_image->GetHeight();
  const int keyframe_width = keyframe_->depth_image->GetWidth();
  const int keyframe_height = keyframe_->depth_image->GetHeight();
  const float* frame_depths = frame.depth_image->GetData();
  const float* keyframe_intensities = keyframe_intensities_.GetData();
  const float* frame_intensities = frame_intensities_.GetData();
  const float* keyframe_depths = keyframe_->depth_image->GetData();
  const float* frame_gradient_x = frame_gradient_x_.GetData();
  const float* frame_gradient_y = frame_gradient_y_.GetData();
  const Vector3f* keyframe_normals = keyframe_->normal_image->GetData();
  const Vector3f* frame_normals = frame.normal_image->GetData();
  const Projection& frame_projection = frame.projection;
  const Projection& keyframe_projection = keyframe_->projection;
  const Transform Tcm = frame.Tcw * keyframe_->Tcw.Inverse();
  float* hessian = hessian_.GetData();
  float* gradient = gradient_.GetData();
  float* residuals = residuals_.GetData();

  thrust::device_ptr<float> dh(hessian);
  thrust::device_ptr<float> dg(gradient);
  thrust::fill(dh, dh + hessian_.GetSize(), 0.0f);
  thrust::fill(dg, dg + gradient_.GetSize(), 0.0f);

  const dim3 threads(16, 16);
  const dim3 total(keyframe_width, keyframe_height);
  const dim3 blocks = GetKernelBlocks(total, threads);

  if (translation_enabled_)
  {
    CUDA_LAUNCH(ComputeSystemKernel<true>, blocks, threads, 0, 0, Tcm,
        keyframe_depths, keyframe_normals, keyframe_intensities,
        keyframe_projection, keyframe_width, keyframe_height, frame_depths,
        frame_normals, frame_intensities, frame_gradient_x, frame_gradient_y,
        frame_projection, frame_width, frame_height, hessian, gradient, residuals);
  }
  else
  {
    CUDA_LAUNCH(ComputeSystemKernel<false>, blocks, threads, 0, 0, Tcm,
        keyframe_depths, keyframe_normals, keyframe_intensities,
        keyframe_projection, keyframe_width, keyframe_height, frame_depths,
        frame_normals, frame_intensities, frame_gradient_x, frame_gradient_y,
        frame_projection, frame_width, frame_height, hessian, gradient, residuals);
  }

  thrust::device_ptr<const float> rptr(residuals);
  thrust::host_vector<float> hptr(rptr, rptr + residuals_.GetSize());
  cv::Mat image(keyframe_height, keyframe_width, CV_32FC1, hptr.data());
  image.convertTo(image, CV_8UC1, 255);
  cv::imwrite("residuals.png", image);

  // ComputeResidual(frame);
  // ComputeJacobian(frame);
  // ComputeHessian();
  // ComputeGradient();
}

} // namespace vulcan