#include <vulcan/tracker.h>
#include <vulcan/device.h>
#include <vulcan/frame.h>
#include <vulcan/projection.h>
#include <vulcan/transform.h>

namespace vulcan
{

void Tracker::ComputeHessian()
{
  const float one = 1;
  const float zero = 0;
  const int residual_count = residuals_.GetSize();
  const int parameter_count = GetParameterCount();
  const float* jacobian = jacobian_.GetData();
  float* hessian = hessian_.GetData();

  CUBLAS_DEBUG(hipblasSgemm(handle_, HIPBLAS_OP_T, HIPBLAS_OP_N, parameter_count,
      parameter_count, residual_count, &one, jacobian, residual_count,
      jacobian, residual_count, &zero, hessian, parameter_count));
}

void Tracker::ComputeGradient()
{
  const float one = 1;
  const float zero = 0;
  const int residual_count = residuals_.GetSize();
  const int parameter_count = GetParameterCount();
  const float* jacobian = jacobian_.GetData();
  const float* residuals = residuals_.GetData();
  float* gradient = gradient_.GetData();

  CUBLAS_DEBUG(hipblasSgemv(handle_, HIPBLAS_OP_T, residual_count,
    parameter_count, &one, jacobian, residual_count, residuals, 1, &zero,
    gradient, 1));
}

} // namespace vulcan