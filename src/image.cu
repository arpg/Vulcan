#include "hip/hip_runtime.h"
#include <vulcan/image.h>
#include <vulcan/device.h>

namespace vulcan
{

template <int BLOCK_DIM>
VULCAN_GLOBAL
void GetGradientsKernel(int width, int height, const float* values,
    float* x_gradients, float* y_gradients)
{
  // allocate shared memory
  const int buffer_dim = BLOCK_DIM + 2;
  const int buffer_size = buffer_dim * buffer_dim;
  VULCAN_SHARED float buffer[buffer_size];

  // get launch indices
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  int sindex = threadIdx.y * blockDim.x + threadIdx.x;
  const int block_size = blockDim.x * blockDim.y;

  // copy image patch to shared memory
  do
  {
    // initialize default value
    float value = 0;

    // get source image indices
    const int vx = (blockIdx.x * blockDim.x - 1) + (sindex % buffer_dim);
    const int vy = (blockIdx.y * blockDim.y - 1) + (sindex / buffer_dim);

    // check if within image bounds
    if (vx >= 0 && vx < width && vy >= 0 && vy < height)
    {
      // read value from global memory
      value = values[vy * width + vx];
    }

    // store value in shared memory
    buffer[sindex] = value;

    // advance to next shared index
    sindex += block_size;
  }
  while (sindex < buffer_size);

  // wait for all threads to finish
  __syncthreads();

  // check if current thread within image bounds
  if (x < width && y < height)
  {
    // initialize default gradients
    float gx = 0;
    float gy = 0;

    // get kernel top-left indices in shared memory
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    // read top row values from shared memory
    const float i00 = 0.125f * buffer[(ty + 0) * buffer_dim + (tx + 0)];
    const float i01 = 0.250f * buffer[(ty + 0) * buffer_dim + (tx + 1)];
    const float i02 = 0.125f * buffer[(ty + 0) * buffer_dim + (tx + 2)];

    // read center row values from shared memory
    const float i10 = 0.250f * buffer[(ty + 1) * buffer_dim + (tx + 0)];
    const float i12 = 0.250f * buffer[(ty + 1) * buffer_dim + (tx + 2)];

    // read bottom row values from shared memory
    const float i20 = 0.125f * buffer[(ty + 2) * buffer_dim + (tx + 0)];
    const float i21 = 0.250f * buffer[(ty + 2) * buffer_dim + (tx + 1)];
    const float i22 = 0.125f * buffer[(ty + 2) * buffer_dim + (tx + 2)];

    // compute gradient values
    gx = (i02 + i12 + i22) - (i00 + i10 + i20);
    gy = (i20 + i21 + i22) - (i00 + i01 + i02);

    // store result in global memory
    const int index = y * width + x;
    x_gradients[index] = gx;
    y_gradients[index] = gy;
  }
}

template <bool nearest>
VULCAN_GLOBAL
void DownsampleKernel(int src_w, int src_h, const float* src,
    int dst_w, int dst_h, float* dst)
{
  const int dst_x = blockIdx.x * blockDim.x + threadIdx.x;
  const int dst_y = blockIdx.y * blockDim.y + threadIdx.y;

  if (dst_x < dst_w && dst_y < dst_h)
  {
    float sample = 0;
    const int src_x = 2 * dst_x;
    const int src_y = 2 * dst_y;

    if (nearest)
    {
      sample = src[src_y * src_w + src_x];
    }
    else
    {
      const int src_x = 2 * dst_x;
      const int src_y = 2 * dst_y;
      sample += src[(src_y + 0) * src_w + (src_x + 1)];
      sample += src[(src_y + 0) * src_w + (src_x + 0)];
      sample += src[(src_y + 1) * src_w + (src_x + 1)];
      sample += src[(src_y + 1) * src_w + (src_x + 0)];
      sample *= 0.25f;
    }

    dst[dst_y * dst_w + dst_x] = sample;
  }
}

template <bool nearest>
VULCAN_GLOBAL
void DownsampleKernel(int src_w, int src_h, const Vector3f* src,
    int dst_w, int dst_h, Vector3f* dst)
{
  const int dst_x = blockIdx.x * blockDim.x + threadIdx.x;
  const int dst_y = blockIdx.y * blockDim.y + threadIdx.y;

  if (dst_x < dst_w && dst_y < dst_h)
  {
    Vector3f sample(0, 0, 0);
    const int src_x = 2 * dst_x;
    const int src_y = 2 * dst_y;

    if (nearest)
    {
      sample = src[src_y * src_w + src_x];
    }
    else
    {
      const int src_x = 2 * dst_x;
      const int src_y = 2 * dst_y;
      sample += src[(src_y + 0) * src_w + (src_x + 1)];
      sample += src[(src_y + 0) * src_w + (src_x + 0)];
      sample += src[(src_y + 1) * src_w + (src_x + 1)];
      sample += src[(src_y + 1) * src_w + (src_x + 0)];
      sample *= 0.25f;
    }

    dst[dst_y * dst_w + dst_x] = sample;
  }
}

void Image::GetGradients(Image& gx, Image& gy) const
{
  gx.Resize(size_);
  gy.Resize(size_);

  float* gx_data = gx.GetData();
  float* gy_data = gy.GetData();

  const dim3 threads(16, 16);
  const dim3 total(size_[0], size_[1]);
  const dim3 blocks = GetKernelBlocks(total, threads);

  CUDA_LAUNCH(GetGradientsKernel<16>, blocks, threads, 0, 0,
      size_[0], size_[1], data_, gx_data, gy_data);
}

void Image::Downsample(Image& image, bool nearest) const
{
  VULCAN_DEBUG_MSG(size_[0] % 2 == 0 && size_[1] % 2 == 0,
      "even image dimensions required");

  image.Resize(size_ / 2);
  const float* src = data_;
  float* dst = image.GetData();

  const int src_w = size_[0];
  const int src_h = size_[1];
  const int dst_w = image.GetWidth();
  const int dst_h = image.GetHeight();

  const dim3 threads(16, 16);
  const dim3 total(dst_w, dst_h);
  const dim3 blocks = GetKernelBlocks(total, threads);

  if (nearest)
  {
    CUDA_LAUNCH(DownsampleKernel<true>, blocks, threads, 0, 0, src_w, src_h,
        src, dst_w, dst_h, dst);
  }
  else
  {
    CUDA_LAUNCH(DownsampleKernel<false>, blocks, threads, 0, 0, src_w, src_h,
        src, dst_w, dst_h, dst);
  }
}

void ColorImage::Downsample(ColorImage& image, bool nearest) const
{
  VULCAN_DEBUG_MSG(size_[0] % 2 == 0 && size_[1] % 2 == 0,
      "even image dimensions required");

  image.Resize(size_ / 2);
  const Vector3f* src = data_;
  Vector3f* dst = image.GetData();

  const int src_w = size_[0];
  const int src_h = size_[1];
  const int dst_w = image.GetWidth();
  const int dst_h = image.GetHeight();

  const dim3 threads(16, 16);
  const dim3 total(dst_w, dst_h);
  const dim3 blocks = GetKernelBlocks(total, threads);

  if (nearest)
  {
    CUDA_LAUNCH(DownsampleKernel<true>, blocks, threads, 0, 0, src_w, src_h,
        src, dst_w, dst_h, dst);
  }
  else
  {
    CUDA_LAUNCH(DownsampleKernel<false>, blocks, threads, 0, 0, src_w, src_h,
        src, dst_w, dst_h, dst);
  }
}

} // namespace vulcan